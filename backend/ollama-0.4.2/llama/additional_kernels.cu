#include <hip/hip_runtime.h>
#include <cmath>

// Kernel to compute the mean vector
__global__ void computeMeanKernel(const float *embeddings, float *mean, size_t embedding_count, size_t embedding_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < embedding_dim) {
        float sum = 0.0f;
        for (size_t i = 0; i < embedding_count; ++i) {
            sum += embeddings[i * embedding_dim + idx];
        }
        mean[idx] = sum;
    }
}

// Kernel to center the data
__global__ void centerDataKernel(const float *embeddings, const float *mean, float *centered_data, size_t embedding_count, size_t embedding_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < embedding_count * embedding_dim) {
        int dim_idx = idx % embedding_dim;
        centered_data[idx] = embeddings[idx] - mean[dim_idx];
    }
}

// Kernel to generate Zipf weights
__global__ void generateZipfWeightsKernel(float *zipf_weights, size_t count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        zipf_weights[idx] = log1p(idx + 1);
    }
}

// Kernel to apply Zipf weights to embeddings
__global__ void applyZipfWeightsKernel(float *embeddings, const float *zipf_weights, size_t embedding_count, size_t embedding_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < embedding_count * embedding_dim) {
        int embedding_idx = idx / embedding_dim;
        embeddings[idx] *= zipf_weights[embedding_idx];
    }
}
